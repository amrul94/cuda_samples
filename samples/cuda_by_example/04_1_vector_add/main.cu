#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include <format>
#include <iostream>

#include "utilities/error_handling.cuh"
#include "utilities/properties.cuh"

__host__ size_t getVectorSize(int argc, char *argv[]) {
  if (argc != 2) {
    std::cerr << "Incorrect number of arguments: " << argc - 1
              << " but expect 1" << std::endl;
    std::exit(EXIT_FAILURE);
  }

  char *end = nullptr;
  size_t ret = strtoul(argv[1], &end, 10);

  if (errno == ERANGE) {
    std::perror("!! Problem is -> ");
    std::exit(EXIT_FAILURE);
  } else if (ret) {
    return ret;
  } else {
    std::cerr << "No number found input is -> " << argv[1] << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

__global__ void vectorAddKernel(const int *a, const int *b, int *c, size_t n) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  while (tid < n) {
    c[tid] = a[tid] + b[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__host__ void vectorAdd(const int *h_a, const int *h_b, int *h_c, size_t n) {
  int *d_a, *d_b, *d_c;
  const size_t size = n * sizeof(int);

  HANDLE_ERROR(hipMalloc((void **)&d_a, size));
  HANDLE_ERROR(hipMalloc((void **)&d_b, size));
  HANDLE_ERROR(hipMalloc((void **)&d_c, size));

  HANDLE_ERROR(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));

  const auto [grid_dim, block_dim] = utils::getGridAndBlockDims(n, 1);
  vectorAddKernel<<<grid_dim, block_dim>>>(d_a, d_b, d_c, n);

  HANDLE_ERROR(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

int main(int argc, char *argv[]) {
  size_t vec_size = getVectorSize(argc, argv);
  std::cout << "Number of elements: " << vec_size << std::endl;

  auto *a = new int[vec_size];
  auto *b = new int[vec_size];
  auto *c = new int[vec_size];

  for (int i = 0; i < vec_size; ++i) {
    a[i] = i;
    b[i] = 2 * i;
  }

  vectorAdd(a, b, c, vec_size);

  bool success = true;
  for (int i = 0; i < vec_size; i++) {
    if ((a[i] + b[i]) != c[i]) {
      printf("Error:  %d + %d != %d\n", a[i], b[i], c[i]);
      success = false;
    }
  }
  if (success) {
    printf("We did it!\n");
  }

  delete[] a;
  delete[] b;
  delete[] c;
}
