#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include "samples/cuda_by_example/common/cpu_bitmap.h"
#include "utilities/error_handling.cuh"

// TODO(amrulla): сделать параметром настраиваемым через командную строку.
constexpr unsigned int DIM = 1024;
constexpr float PI = 3.1415926535897932f;

__device__ float partialCalc(unsigned int v, float period) {
  const auto fv = static_cast<float>(v);
  return sinf(fv * 2.0f * PI / period) + 1.0f;
}

__global__ void kernel(unsigned char *ptr) {
  unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int offset = x + y * blockDim.x * gridDim.x;

  __shared__ float shared[16][16];

  const float period = 128.0f;

  const float vx = partialCalc(x, period);
  const float vy = partialCalc(y, period);
  shared[threadIdx.x][threadIdx.y] = 255 * vx * vy / 4.0f;

  __syncthreads();

  const float color = shared[15 - threadIdx.x][15 - threadIdx.y];

  ptr[offset * 4 + 0] = 0;
  ptr[offset * 4 + 1] = static_cast<unsigned char>(color);
  ptr[offset * 4 + 2] = 0;
  ptr[offset * 4 + 3] = 255;
}

int main() {
  CPUBitmap bitmap{DIM, DIM};
  unsigned char *dev_bitmap;

  HANDLE_ERROR(hipMalloc((void **)&dev_bitmap, bitmap.image_size()));

  dim3 grids{DIM / 16, DIM / 16};
  dim3 threads{16, 16};

  kernel<<<grids, threads>>>(dev_bitmap);

  HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(),
                          hipMemcpyDeviceToHost));

  bitmap.display_and_exit();

  HANDLE_ERROR(hipFree(dev_bitmap));
}
